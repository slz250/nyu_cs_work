#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//function declarations
void populateArr(unsigned int* num);
unsigned int getmax(unsigned int* num, unsigned int size);
//

void populateArr(unsigned int* num, unsigned int size) {
  unsigned int i;
  srand(time(NULL)); // setting a seed for the random number generator
  // Fill-up te array with random num from 0 to size-1 
  for(i = 0; i < size; i++) {
    num[i] = rand() % size;
  }
}

/*
input: pointer to an array of unsigned int number of elements in the array
output: the maximum number of the array
*/
unsigned int getmax(unsigned int* num, unsigned int size) {
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++) {
    if(num[i] > max) {
      max = num[i];
    }
  }
  return (max);
}

__global__ void getmaxcu(unsigned int* numD, unsigned int numThreads) {
  unsigned int i , stride;
  unsigned int uniqueIndex = threadIdx.x + (blockDim.x * blockIdx.x);  

  stride = numThreads / 10;
  if (uniqueIndex < stride) {
    for (i = 1; i < 10; i++) {
      if (numD[uniqueIndex] < numD[uniqueIndex + stride * i]) {
        numD[uniqueIndex] = numD[uniqueIndex + stride * i];
      }
      __syncthreads();
    }
  }
  __syncthreads();
}

int main(int argc, char *argv[]) {
  unsigned int size = 0;  // The size of the array
  unsigned int* num; //pointer to the array

  if(argc !=2) {
    printf("usage: maxseq num\n");
    printf("num = size of the array\n");
    exit(1);
  }

  size = atol(argv[1]);

  num = (unsigned int *) malloc(size * sizeof(unsigned int));
  if(!num) {
    printf("Unable to allocate mem for an array of size %u\n", size);
    exit(1);
  }    

  populateArr(num, size);

  // printf("Ser max: %u\n", getmax(num, size));
  unsigned int threadsPerBlock = 1024;
  unsigned int numBlocks = (unsigned int) ceil((double) size/threadsPerBlock);
  unsigned int dataSize = size * sizeof(unsigned int);
  unsigned int* numD;

  hipMalloc((void**) &numD, dataSize);
  hipMemcpy(numD, num, dataSize, hipMemcpyHostToDevice);

  unsigned int numThreads;

  for (numThreads = size; numThreads > 1; numThreads = numThreads / 10) {
    getmaxcu<<<numBlocks, threadsPerBlock>>>(numD, numThreads);
  }

  hipMemcpy(num, numD, dataSize, hipMemcpyDeviceToHost);

  printf("Par max: %u\n", num[0]);


  free(num);
  hipFree(numD);
  exit(0);
}